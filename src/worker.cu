#include "hip/hip_runtime.h"
/*
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <stdint.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <pthread.h>
#include <semaphore.h>
#include <zmq.h>
#include <czmq.h>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>
#include "util.h"
#include "sha1.h"
#include "rsa.h"
#include "x509.h"
#include "dkey.h"
#include "dkim.h"

/* Structure to be passed between the CPU and GPU, conveying the results of the signing process */
typedef struct metastore
{
  uint32_t used;
  uint32_t length[TOTALMESSAGES];
  uint32_t offset[TOTALMESSAGES];

  char taskid[TOTALMESSAGES][SMALLSTRLEN];
  uint8_t signatures[TOTALMESSAGES];
  uint8_t sigtype[TOTALMESSAGES][2];
  char signature[TOTALMESSAGES][2][SMALLSTRLEN];

} metastore;

/* Structure to keep track of a CUDA stream and its associated message blocks */
typedef struct streamstore
{
  uint32_t device;
  uint32_t streamid;
  char *host_message_mem;
  metastore *host_meta_mem;
  char *card_message_mem;
  metastore *card_meta_mem;
  hipStream_t stream;
  sem_t writesem;
} streamstore;

streamstore * streams;            /* Pointer to all streams */
uint32_t totalstreams = 0;        /* Number of streams based on number of devices */
uint32_t curstream = 0;            /* Index of stream being written */
pthread_mutex_t transferlock;    /* Only allow one thread to change the current stream */

int workertype = 0;                /* CPU=0 or GPU=1 */
char nodeid[SMALLSTRLEN];            /* Worker identity passed on command line */
char controller[SMALLSTRLEN];        /* ZeroMQ path for communicating with controller node */

FILE * logfileh = stdout;        /* Handle for writing log messages */
int threadcount = 0;            /* Number of signing threads to run (CPU nodes only) */
int devicecount = 0;            /* Numner of CUDA devices available */
queue ** waitingqueues = NULL;    /* Queues to hold messages waiting to be signed */
queue * controllerqueue = NULL;    /* Queue to hold messages for sending to the controller */
uint32_t apprunning = 1;        /* Global status to track whether application should still be running */
time_t lastkeepalive = 0;        /* Time that keepalive message was last sent to controller */

/* Function to message to the controller node to signify that the worker is still running */
/* Messages are sent at a maximum frequency of once a second */
int send_keepalive(char * nodeid, int workertype, void * socket)
{
  if((time(NULL) - lastkeepalive) > 0)
  {
    lastkeepalive = time(NULL);

    /* Build task from arguments */
    uint32_t total_size = 0, offset = 0;
    uint8_t message_type = TYPE_KEEPALIVE_REQUEST;
    uint8_t nodeid_len = strlen(nodeid) + 1;

    total_size = total_size + sizeof(uint8_t);                    /* Account for message type value */
    total_size = total_size + sizeof(uint8_t) + nodeid_len;     /* Account for node id length and value */
    total_size = total_size + sizeof(uint8_t);                    /* Account for node type value */
    total_size = total_size + sizeof(uint8_t);                    /* Account for thread or device count */

    zmq_msg_t message;
    zmq_msg_init_size (&message, total_size);
    char * msg_data = (char *)zmq_msg_data(&message);

    /* Field length = 4, Field value = 0 (Register) */
    write_uint8_value(msg_data, &offset, total_size, message_type);

    /* Node ID */
    write_uint8_length_value(msg_data, &offset, nodeid_len, total_size, nodeid);

    /* Worker type */
    write_uint8_value(msg_data, &offset, total_size, workertype);

    /* Thread or device */
    write_uint8_value(msg_data, &offset, total_size, workertype ? devicecount: threadcount);

    /* Send message */
    int size_sent = zmq_msg_send (&message, socket, 0);
    if(size_sent == -1)
    {
      host_log_printf(logfileh, "Error: Failed to send keepalive message to controller\n");
      return 1;
    }
    zmq_msg_close (&message);

    if(size_sent != total_size)
    {
      host_log_printf(logfileh, "Error: Failed to send message to controller\n");
      return 1;
    }
  }

  return 0;
}

/* Callback function that is run after a stream of signing requests is complete */
void stream_callback(hipStream_t stream, hipError_t status, void *data)
{
  if(apprunning == 0) return;

  streamstore * ss = (streamstore *)data;
  metastore * hms = ss->host_meta_mem;

  host_log_printf(logfileh, "Info: Sending responses from stream %i...\n", ss->streamid);

  /* Build a signing response message for each task and put onto the controller queue */
  int resi;
  for(resi = 0; resi < hms->used; resi++)
  {
    task * taskresponse = (task *)calloc(1, sizeof(struct task));
    if(taskresponse == NULL)
    {
      host_log_printf(logfileh, "Error: Unable to allocate memory for task\n");
      return;
    }

    taskresponse->type = TYPE_SIGN_RESPONSE;
    strcpy(taskresponse->id, hms->taskid[resi]);
    taskresponse->id_len = strlen(taskresponse->id) + 1;

    /* Populate signatures from returned results */
    int sigi;
    for(sigi = 0; sigi < hms->signatures[resi]; sigi++)
    {
      if((hms->sigtype[resi][sigi] & DKIM_MODE) != 0)
      {
        strcpy(taskresponse->dkim, hms->signature[resi][sigi]);
        taskresponse->dkim_len = strlen(taskresponse->dkim) + 1;
      }
      if((hms->sigtype[resi][sigi] & DKEY_MODE) != 0)
      {
        strcpy(taskresponse->dkey, hms->signature[resi][sigi]);
        taskresponse->dkey_len = strlen(taskresponse->dkey) + 1;
      }
    }

    add_to_queue(controllerqueue, taskresponse);
  }

  /* Clear host memory for next use */
  memset(ss->host_meta_mem, 0, sizeof(struct metastore));
}

/* Kernel function to be run on CUDA capable card */
/* Function processes one message, producing DKEY and/or DKIM signatures depending on request */
__global__ void signmessagekernel(void * messagedatain, void * metadatain, time_t signtime)
{
  /* Find message to be signed based on block and thread IDs */
  metastore * metadata = (metastore *)metadatain;
  uint32_t index = (blockIdx.x * MESSAGESPERBLOCK) + threadIdx.x;

  /* Check there is a message waiting in the area allocated to this thread */
  if(index < metadata->used)
  {
    char * messagedata = (char *)messagedatain + metadata->offset[index];

    task newtaskp;
    memset(&newtaskp, 0, sizeof(struct task));

    uint32_t offset = 0;

    /* Populate task structure from raw message block */
    if(read_sign_request(messagedata, &offset, metadata->length[index], &newtaskp) != 0)
    {
      device_log_printf("Error: Failed to read signing request\n");
      return;
    }

    /* Set attributes for response */
    if(cuda_strncpy(metadata->taskid[index], 256, newtaskp.id) != 0) return;

    /* Signature timestamp */
    newtaskp.ts = signtime;

    if((newtaskp.mode & DKIM_MODE) != 0)
    {
      /* Create DKIM signature */
      if(dkim_create(&newtaskp) != 0)
      {
        device_log_printf("Error: Failed to create DKIM signature\n");
        return;
      }

      /* Populate returning metadata block with calculated signature */
      metadata->sigtype[index][metadata->signatures[threadIdx.x]] += DKIM_MODE;
      cuda_strcpy((char *)(metadata->signature[index][metadata->signatures[index]]), newtaskp.dkim);
      metadata->signatures[index]++;
    }

    if((newtaskp.mode & DKEY_MODE) != 0)
    {
      /* Create DKEY signature */
      if(dkey_create(&newtaskp) != 0)
      {
        device_log_printf("Error: Failed to create DKEY signature\n");
        return;
      }

      /* Populate returning metadata block with calculated signature */
      metadata->sigtype[index][metadata->signatures[index]] += DKEY_MODE;
      cuda_strcpy((char *)(metadata->signature[index][metadata->signatures[index]]), newtaskp.dkey);
      metadata->signatures[index]++;
    }
  }
}

/**
* This macro checks return value of the CUDA runtime call and exits
* the application if the call failed.
*/
#define CUDA_CHECK_RETURN(value) {                                              \
hipError_t _m_cudaStat = value;                                                \
if (_m_cudaStat != hipSuccess) {                                               \
  host_log_printf(logfileh, "Error: %s at line %d in file %s\n",                \
    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);                       \
  return 1;                                                                     \
} }

/* Function called whenever there is a full block of messages ready to be deployed to the CUDA card */
/* A block is full when either the maxmimum number of messages is populated, or the culmulative size of the messages reaches the maximum block size */
int send_block(streamstore * ss, metastore * hms)
{
  host_log_printf(logfileh, "Info: Signing stream %i on device %i with %i messages...\n", curstream, ss->device, hms->used);

  hipSetDevice(ss->device);

  /* Allocate space for tasks on card and copy over */
  CUDA_CHECK_RETURN(hipMalloc((void**) &(ss->card_message_mem), MESSAGESIZE * TOTALMESSAGES));
  CUDA_CHECK_RETURN(hipMemcpyAsync(ss->card_message_mem, ss->host_message_mem, MESSAGESIZE * TOTALMESSAGES, hipMemcpyHostToDevice, ss->stream));

  /* Allocate space for metadata on card and copy over */
  CUDA_CHECK_RETURN(hipMalloc((void**) &(ss->card_meta_mem), sizeof(struct metastore)));
  CUDA_CHECK_RETURN(hipMemcpyAsync(ss->card_meta_mem, ss->host_meta_mem, sizeof(struct metastore), hipMemcpyHostToDevice, ss->stream));

  /* Run the kernel */
  signmessagekernel<<<NUMBEROFBLOCKS, MESSAGESPERBLOCK, 0, ss->stream>>>(ss->card_message_mem, ss->card_meta_mem, time(NULL));

  /* Copy back metadata */
  CUDA_CHECK_RETURN(hipMemcpyAsync(ss->host_meta_mem, ss->card_meta_mem, sizeof(struct metastore), hipMemcpyDeviceToHost, ss->stream));

  /* Free space used on card */
  CUDA_CHECK_RETURN(hipFree((void*) ss->card_message_mem));
  CUDA_CHECK_RETURN(hipFree((void*) ss->card_meta_mem));

  /* Add callback for after all these asynchronous actions have been completed */
  hipStreamAddCallback(ss->stream, stream_callback, (void *)ss, 0);

  /* Move to next stream */
  curstream = (curstream + 1)%totalstreams;
  streams[curstream].host_meta_mem->used = 0;

  return 0;
}

/* Function to add the raw content of a request into the memory block to be transferred to the CUDA card */
/* Checks for space, transmitting current block and moving to the next block if necessary */
int add_to_transfer(zframe_t * content, uint32_t offset)
{
  pthread_mutex_lock(&transferlock);

  streamstore * ss = &(streams[curstream]);
  metastore * hms = ss->host_meta_mem;

  /* add_to_transfer is called without content in order to force sending of the block (timeout) */
  if(content == NULL)
  {
    /* Check there is actually something to send */
    if(hms->used != 0)
    {
      /* Force send */
      send_block(ss, hms);
    }
  } else {

    /* Calculate position for next message */
    if(hms->used != 0)
    {
      hms->offset[hms->used] = hms->offset[hms->used - 1] + hms->length[hms->used - 1];

      /* Check for space in current stream block */
      if(((hms->offset[hms->used] + zframe_size(content)) >= (MESSAGESPERBLOCK * MESSAGESIZE)) || (hms->used == MESSAGESPERBLOCK))
      {
        send_block(ss, hms);
      }
    }
  }

  /* Add incoming message to transfer block */
  if(content != NULL)
  {
    /* Set stream in the block */
    streamstore * ss = &streams[curstream];
    metastore * hms = ss->host_meta_mem;
    ss->streamid = curstream;

    /* Copy data */
    char * framedata = (char *)zframe_data(content);
    hms->length[hms->used] = zframe_size(content) - offset;
    memcpy(ss->host_message_mem + hms->offset[hms->used], framedata + offset, hms->length[hms->used]);

    hms->used++;

    /* Check if the block is full */
    if(hms->used == MESSAGESPERBLOCK)
    {
      send_block(ss, hms);
    }

    /* Indicate progress to prevent timeout */
    sem_post(&(streams[curstream].writesem));
  }

  pthread_mutex_unlock(&transferlock);

  return 0;
}

/* Extract details of signing request from the controller */
int process_controller_request(zframe_t * content, int threadnum)
{
  /* Create a task structure to carry the details of this request */
  task * newtask = (task *)calloc(1, sizeof(task));
  if(newtask == NULL)
  {
    host_log_printf(logfileh, "Error: Unable to allocate memory for task\n");
    return 1;
  }
  memset(newtask, 0, sizeof(struct task));

  /* Parse request from client */
  uint32_t offset = 0;
  char * framedata = (char *)zframe_data(content);

  /* Check message size is within allowable limit */
  if(zframe_size(content) > ((TOTALMESSAGES-1) * MESSAGESIZE))
  {
    host_log_printf(logfileh, "Error: Message is larger then maximum permitted message size\n");
    zframe_destroy(&content);
    free(newtask);
    return 1;
  }

  /* Read message type */
  if(read_uint8_value(framedata, &offset, zframe_size(content), &(newtask->type)) != 0)
  {
    host_log_printf(logfileh, "Error: Failed to read type of request received from controller\n");
    zframe_destroy(&content);
    free(newtask);
    return 1;
  }

  switch(newtask->type)
  {
  case TYPE_SIGN_REQUEST:
  {
    if(workertype == 0)
  {
    /* CPU based node - Extract task details and add to queue */
    newtask->content = content;
    if(read_sign_request((char *)zframe_data(newtask->content), &offset, zframe_size(content), newtask) != 0)
    {
      host_log_printf(logfileh, "Error: Failed to read task details\n");
      zframe_destroy(&content);
      free(newtask);
      break;
    }

    /* Add task to queue of next available thread */
    add_to_queue(waitingqueues[threadnum], newtask);

  } else {
    /* GPU based - write into memory block for transfer */
    add_to_transfer(content, offset);
    zframe_destroy(&content);
    free(newtask);
  }

  break;
  }
  default:
  {
    host_log_printf(logfileh, "Error: Unknown message type\n");
    zframe_destroy(&content);
    free(newtask);
    break;
  }
  }

  return 0;
}

/* CPU based thread for signing messages */
/* Reads requests from incoming queue, processes, and pass back to controller */
void * signmessagethread(void * param)
{
  queue * waitingqueue = (queue *)param;

  while(apprunning)
  {
    /* Wait for something to send */
    if(sem_trywait(&(waitingqueue->sendsem)) == 0)
    {
      while(waitingqueue->head != NULL)
      {
        /* Read task from front of queue */
        task * taskp = waitingqueue->head;

        /* Set timestamp for message */
        taskp->ts = time(NULL);

        if((taskp->mode & DKEY_MODE) != 0)
        {
          /* Generate DKEY signature */
          if(dkey_create(taskp) != 0)
          {
            taskp->dkey[0] = '\0';
            taskp->dkey_len = 0;
            host_log_printf(logfileh, "Error: Unable to create DKEY signature\n");
          }
        }

        if((taskp->mode & DKIM_MODE) != 0)
        {
          /* Generate DKIM signature */
          if(dkim_create(taskp) != 0)
          {
            taskp->dkim[0] = '\0';
            taskp->dkim_len = 0;
            host_log_printf(logfileh, "Error: Unable to create DKIM signature\n");
          }
        }

        /* Update message type so that it will get sent back */
        taskp->type = TYPE_SIGN_RESPONSE;

        /* Return to controller */
        move_task(taskp, waitingqueue, controllerqueue);
      }
    } else {
      usleep(10000);
    }
  }

  return NULL;
}

/* GPU based thread for signing messages */
/* Monitors activity and forces send of outstanding messages to card after inactivity */
void * gpusignmessagethread(void * param)
{
  struct timespec delay;
  delay.tv_nsec = 0;
  int writestream = 0;

  while(apprunning)
  {
    /* Wait for something to send */
    delay.tv_sec = time(NULL) + 30;
    if(sem_timedwait(&(streams[writestream].writesem), &delay) == 0)
    {
      /* Message has been written during wait - Don't need to do anything */
    } else {
      /* Timeout has occurred - force send */
      add_to_transfer(NULL, 0);
    }
  }

  return NULL;
}

/* Main thread for communicating with controller */
void * controllerthread(void * param)
{
  void * context;
  void * cc_socket;

  /* Create socket for sending messages */
  context = zmq_ctx_new ();
  if(context == NULL)
  {
    host_log_printf(logfileh, "Error: Unable to create zmq context\n");
    return NULL;
  }

  cc_socket = zmq_socket (context, ZMQ_DEALER);

  if(cc_socket == NULL)
  {
    host_log_printf(logfileh, "Error: Unable to open zmq socket\n");
    return NULL;
  }

  /* Set socket identity based on node id passed on command line */
  char * idstring = (char *)param;
  //zsockopt_set_identity (cc_socket, idstring);

  /* Don't hold on to messages if the other end drops */
  int timeout = 0;
  if(zmq_setsockopt(cc_socket, ZMQ_LINGER, &timeout, sizeof(timeout)) != 0)
  {
    host_log_printf(logfileh, "Error: Failed to set linger period\n");
    return NULL;
  }

  /* Set large queue sizes to buffer messages - argument is number of messages */
  int qsize = 100000;
  if(zmq_setsockopt(cc_socket, ZMQ_SNDHWM, &qsize, sizeof(qsize)) != 0)
  {
    host_log_printf(logfileh, "Error: Failed to set queue size\n");
    return NULL;
  }

  if(zmq_setsockopt(cc_socket, ZMQ_RCVHWM, &qsize, sizeof(qsize)) != 0)
  {
    host_log_printf(logfileh, "Error: Failed to set queue size\n");
    return NULL;
  }

  /* Establish connection to controller */
  if(zmq_connect (cc_socket, controller) != 0)
  {
    host_log_printf(logfileh, "Error: Failed to connect to listening socket\n");
    return NULL;
  }

  int thread_to_use = 0;
  zmq_pollitem_t items [] = { { cc_socket, 0, ZMQ_POLLIN, 0 } };
  while (apprunning)
  {
    /* Send update to controller to show availability */
    send_keepalive(nodeid, workertype, cc_socket);

    /* Check of any incoming messages */
    int numitems = zmq_poll (items, 1, 0);
    switch(numitems)
    {
    case -1:
    {
      host_log_printf(logfileh, "Error: Failure to poll\n");
      break;
    }
    case 0:
    {
      usleep(10);
      break;
    }
    default:
    {
      if (items[0].revents & ZMQ_POLLIN)
    {
      /* Attempt to read message */
      zmsg_t *msg = zmsg_recv (cc_socket);
      zframe_t *content = zmsg_pop (msg);
      if(content == NULL)
      {
        host_log_printf(logfileh, "Error: Failed to read content of message\n");
        zmsg_destroy (&msg);
        continue;
      }

      /* Select thread to use - round robin */
      if(threadcount != 0)
      {
        thread_to_use = (thread_to_use + 1)%threadcount;
      }

      /* Actual message processing */
      if(process_controller_request(content, thread_to_use) != 0)
      {
        host_log_printf(logfileh, "Error: Failed to process message from controller\n");
        zmsg_destroy (&msg);
        continue;
      }

      zmsg_destroy (&msg);
    }
      break;
    }
    }

    /* Check it there is something to send */
    if(sem_trywait(&(controllerqueue->sendsem)) == 0)
    {
      /* Loop through all messages waiting on the queue */
      while(controllerqueue->head != NULL)
      {
        task * taskp = controllerqueue->head;

        switch(taskp->type)
        {
        case TYPE_SIGN_RESPONSE:
        {
          /* Calculate size required for response message */
          uint32_t total_size = 0, offset = 0, signatures = 0;
          total_size = total_size + sizeof(uint8_t);                     /* Account for message type */
          total_size = total_size + sizeof(uint8_t) + taskp->id_len;     /* Account for task id */
          total_size = total_size + sizeof(uint8_t);                    /* Account for number of signatures */

          if(taskp->dkey_len > 0)
          {
            total_size = total_size + sizeof(uint8_t) + 1;                         /* Account for signature type */
          total_size = total_size + sizeof(uint16_t) + taskp->dkey_len + 1;      /* Account for signature length and value */
          signatures++;
          }

          if(taskp->dkim_len > 0)
          {
            total_size = total_size + sizeof(uint8_t) + 1;                         /* Account for signature type */
          total_size = total_size + sizeof(uint16_t) + taskp->dkim_len + 1;      /* Account for signature length and value */
          signatures++;
          }

          /* Build response message */
          zmq_msg_t message;
          memset(&message, 0, sizeof(zmq_msg_t));
          zmq_msg_init_size (&message, total_size);
          char * msg_data = (char *)zmq_msg_data(&message);
          write_uint8_value(msg_data, &offset, total_size, taskp->type);
          write_uint8_length_value(msg_data, &offset, taskp->id_len, total_size, taskp->id);
          write_uint8_value(msg_data, &offset, total_size, signatures);

          if(taskp->dkey_len > 0)
          {
            write_uint8_value(msg_data, &offset, total_size, DKEY_MODE);
            write_uint16_length_value(msg_data, &offset, taskp->dkey_len, total_size, taskp->dkey);
          }

          if(taskp->dkim_len > 0)
          {
            write_uint8_value(msg_data, &offset, total_size, DKIM_MODE);
            write_uint16_length_value(msg_data, &offset, taskp->dkim_len, total_size, taskp->dkim);
          }

          /* Send response */
          int size_sent = zmq_msg_send (&message, cc_socket, 0);
          if(size_sent == -1)
          {
            host_log_printf(logfileh, "Error: Failed to send message to controller\n");
            break;
          }
          zmq_msg_close (&message);

          if(size_sent != total_size)
          {
            host_log_printf(logfileh, "Error: Failed to send message to controller\n");
            break;
          }

          break;
        }

        default:
        {
          host_log_printf(logfileh, "Error: Request to send unrecognised message type\n");
          break;
        }
        }

        /* Remove message from queue */
        if(remove_from_queue(controllerqueue, taskp) == 0)
        {
          destroy_task(taskp);
        }
      }
    }
  }

  /* Cleanup everything if the application is closing */
  host_log_printf(logfileh, "Cleaning up sockets\n");

  zmq_close (cc_socket);
  zmq_ctx_destroy (context);

  return 0;
}

void termination_handler (int signum)
{
  apprunning = 0;
}

int main(int argc, char **argv)
{
  pthread_t srthread;
  pthread_t * signingthreads = NULL;
  int threadi;

  /* Check command line arguments */
  if((argc != 5) && (argc != 6))
  {
    logfileh = stdout;
    host_log_printf(logfileh, "Usage: worker <Type> <Worker ID> <Number of threads> <Controller location> [<Logfile>]\n");
    host_log_printf(logfileh, "               <Type> = CPU or GPU\n");
    host_log_printf(logfileh, "               <Worker ID> = Unique string to represent this worker instance\n");
    host_log_printf(logfileh, "               <Number of threads> = Concurrent signing threads to run (Relevant to CPU instances only)\n");
    host_log_printf(logfileh, "               <Controller location> = tcp://Address:Port of controlling host\n");
    host_log_printf(logfileh, "               <Log file> = Location for application logging (Console output is used if a logfile is not specified)\n");
    return 1;
  }

  /* Setup signal handling to cleanly shutdown */
  struct sigaction new_action;
  new_action.sa_handler = termination_handler;
  sigemptyset (&new_action.sa_mask);
  new_action.sa_flags = 0;
  sigaction (SIGINT, &new_action, NULL);

  pthread_mutex_init(&transferlock, NULL);

  /* Copy command line arguments that need to be visible globally */
  sprintf(nodeid, "%s-%lu", argv[2], time(NULL));
  strcpy(controller, argv[4]);

  /* Open logfile if specified */
  if(argc == 6)
  {
    logfileh = fopen(argv[5], "a");
    if(logfileh == NULL)
    {
      /* Default to stdout if logfile was not opened */
      logfileh = stdout;
      host_log_printf(logfileh, "Error: Unable to open log file '%s'. Outputting log messages to console.\n", argv[5]);
    }
  }

  /* Check what type of node we are supposed to be running */
  if(strcmp(argv[1], "GPU") == 0)
  {
    /* Check if we have any CUDA devices */
    hipGetDeviceCount(&devicecount);
    host_log_printf(logfileh, "Info: %i CUDA devices detected\n", devicecount);

    /* Calculate memory space required for streams */
    totalstreams = STREAMSPERDEVICE * devicecount;
    streams = (streamstore *)calloc(totalstreams, sizeof(struct streamstore));
    if(streams == NULL)
    {
      host_log_printf(logfileh, "Error: Unable to allocate host memory for streams\n");
      return 1;
    }

    if(devicecount != 0)
    {
      /* Initialise a GPU based worker node */
      host_log_printf(logfileh, "Info: Initialising node as GPU worker\n");
      workertype = 1;
      threadcount = 1;

      /* Loop through all devices */
      int device;
      for (device = 0; device < devicecount; ++device)
      {
        /* Select and reset device */
        hipSetDevice(device);
        hipDeviceReset();

        /* Display device properties */
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        host_log_printf(logfileh, "Info: Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);

        /* Allocate and initialise space for raw message data and metadata */
        for (int i = 0; i < STREAMSPERDEVICE; i++)
        {
          streams[i].device = device;

          streams[i].host_message_mem = (char *)calloc(MESSAGESIZE * TOTALMESSAGES, sizeof(char));
          if(streams[i].host_message_mem == NULL)
          {
            host_log_printf(logfileh, "Error: Unable to allocate host memory for messages\n");
            return 1;
          }

          streams[i].host_meta_mem = (metastore *)calloc(1, sizeof(struct metastore));
          if(streams[i].host_meta_mem == NULL)
          {
            host_log_printf(logfileh, "Error: Unable to allocate host memory for metadata\n");
            return 1;
          }

          streams[i].host_meta_mem->used = 0;
          hipStreamCreate(&(streams[i].stream));

          if(sem_init(&(streams[i].writesem), 0, 0) == -1)
          {
            host_log_printf(logfileh, "Error: Failed to initialise semaphore\n");
            return 1;
          }
        }
      }

      /* Start thread to monitor memory blocks */
      signingthreads = (pthread_t *)calloc(threadcount, sizeof(pthread_t));
      if(signingthreads == NULL)
      {
        host_log_printf(logfileh, "Error: Unable to allocate memory for threads\n");
        return 1;
      }

      pthread_create(&(signingthreads[0]), NULL, gpusignmessagethread, NULL);
    }
  }

  /* Run as a CPU node if GPU was not requested or was not initialised successfully */
  if(workertype == 0)
  {
    /* Initialise a CPU based worker node */
    host_log_printf(logfileh, "Info: Initialising node as CPU worker\n");
    workertype = 0;

    /* Check that a sensible number of threads has been requested */
    threadcount = atoi(argv[3]);
    if((threadcount < 1) || (threadcount > 64))
    {
      threadcount = 1;
    }

    /* Create a queue and a thread for each signing instance */
    waitingqueues = (queue **)calloc(threadcount, sizeof(queue *));
    if(waitingqueues == NULL)
    {
      host_log_printf(logfileh, "Error: Unable to allocate memory for threads\n");
      return 1;
    }

    signingthreads = (pthread_t *)calloc(threadcount, sizeof(pthread_t));
    if(signingthreads == NULL)
    {
      host_log_printf(logfileh, "Error: Unable to allocate memory for threads\n");
      return 1;
    }

    for(threadi = 0; threadi < threadcount; threadi++)
    {
      waitingqueues[threadi] = create_queue(logfileh, threadi);
      pthread_create(&(signingthreads[threadi]), NULL, signmessagethread, waitingqueues[threadi]);
    }
  }

  /* Start one thread for communicating with the controller */
  controllerqueue = create_queue(logfileh, 0);
  pthread_create(&srthread, NULL, controllerthread, nodeid);

  while(apprunning)
  {
    sleep(1);
  }

  /* Cleanup CPU resources when we are shutting down */
  for(threadi = 0; threadi < threadcount; threadi++)
  {
    pthread_join(signingthreads[threadi], NULL);
  }
  free(signingthreads);

  pthread_join(srthread, NULL);

  if(workertype == 0)
  {
    for(threadi = 0; threadi < threadcount; threadi++)
    {
      destroy_queue(waitingqueues[threadi]);
    }
    free(waitingqueues);
  }

  if(workertype == 1)
  {
    /* Cleanup GPU resources */
    int i;
    for (i = 0; i < totalstreams; i++)
    {
      free(streams[i].host_message_mem);
      free(streams[i].host_meta_mem);
      hipSetDevice(streams[i].device);
      hipStreamDestroy(streams[i].stream);
    }
    free(streams);
  }

  destroy_queue(controllerqueue);

  return 0;
}
